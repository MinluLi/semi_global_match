#include "hip/hip_runtime.h"
#include "CTensor.h"
#include "timer.h"

#include <cmath>
#include <sstream>
#include <iostream>
#include <iomanip>


// Convert integer to std::string
template <typename T>
  std::string NumberToString ( T Number )
  {
    std::ostringstream ss;
    ss << Number;
    return ss.str();
  }

/*-------------------------------------------------------------------------
 *  32Bit RGBA color
 *-------------------------------------------------------------------------*/ 
typedef uchar4 Color;

/*-------------------------------------------------------------------------
 *  Regularization weight
 *-------------------------------------------------------------------------*/
static float const LAMBDA = 100.0f;

/*-------------------------------------------------------------------------
 *  Maximum disparity (number of labels in the message passing algorithm)
 *-------------------------------------------------------------------------*/ 
static int const MAX_DISPARITY = 50;

/*======================================================================*/
/*! 
 *   Convert CTensor to CMatrix of Colors.
 *
 *   \param out The output CMatrix
 *   \param in  The input CTensor
 */
/*======================================================================*/
void CTensorToColorCMatrix(
    CMatrix<Color>& out, const CTensor<unsigned char>& in)
{
  out.setSize(in.xSize(), in.ySize());
  for( int y = 0; y < out.ySize(); ++y )
  for( int x = 0; x < out.xSize(); ++x )
  {
    out(x,y).x = in(x,y,0); // R
    out(x,y).y = in(x,y,1); // G
    out(x,y).z = in(x,y,2); // B
  }
}


/*======================================================================*/
/*! 
 *   Compute squared distance of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L2-distance squared of a and b
 */
/*======================================================================*/
inline float unaryL2Squared(Color const &a, Color const &b)
{
  return (static_cast<float>(a.x) - static_cast<float>(b.x)) *
         (static_cast<float>(a.x) - static_cast<float>(b.x)) +
         (static_cast<float>(a.y) - static_cast<float>(b.y)) *
         (static_cast<float>(a.y) - static_cast<float>(b.y)) +
         (static_cast<float>(a.z) - static_cast<float>(b.z)) *
         (static_cast<float>(a.z) - static_cast<float>(b.z));
}


/*======================================================================*/
/*! 
 *   Compute absolute difference of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L1-distance of a and b
 */
/*======================================================================*/
inline float unaryL1(Color const &a, Color const &b)
{
  return abs(static_cast<float>(a.x) - static_cast<float>(b.x)) +
         abs(static_cast<float>(a.y) - static_cast<float>(b.y)) +
         abs(static_cast<float>(a.z) - static_cast<float>(b.z));
}


/*======================================================================*/
/*! 
 *   Compute euclidean L2 distance of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L2-distance of a and b
 */
/*======================================================================*/
inline float unaryEuclidean(Color const &a, Color const &b)
{
 return std::sqrt(unaryL2Squared(a, b));
}


/*======================================================================*/
/*! 
 *   Compute euclidean Lx-norm for pixel neighborhood 
 *
 *   \param leftImg   Left image
 *   \param rightImg  Right image
 *   \param xl        x position of left image pixel
 *   \param yl        y position of left image pixel
 *   \param xr        x position of right image pixel
 *   \param yr        y position of right image pixel
 *   \param costFunction {L1; L2} 
 *   \param N         NxN neighborhood
 *   \param option    1: L1-norm; 2: L2-norm squared
 *
 *   \return Lx-norm for pixel neighborhood
 */
/*======================================================================*/
inline float unaryLxNeighbor(CMatrix<Color> const &leftImg,
                             CMatrix<Color> const &rightImg,
                             int xl, int yl, int xr, int yr,
                             int costFunction,
                             int N)
{
  float theta = 0.0f;
  int lim = static_cast<int>(N/2);
  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (xl+i > 0 && xl+i < leftImg.xSize() &&
          yl+j > 0 && yl+j < leftImg.ySize() &&
          xr+i > 0 && xr+i < rightImg.xSize() &&
          yr+j > 0 && yr+j < rightImg.ySize()) {
        switch(costFunction) {
          case 1:
            theta += unaryL1(leftImg(xl+i, yl+j),
                             rightImg(xr+i, yr+j));
            break;
          case 2:
            theta += unaryL2Squared(leftImg(xl+i, yl+j),
                                    rightImg(xr+i, yr+j));
            break;
        }
      }
    }
  }

  return theta;
}


/*======================================================================*/
/*! 
 *   Compute Average pixel of NxN neighborhood 
 *
 *   \param Img       Image
 *   \param x         x position of image pixel
 *   \param y         y position of image pixel
 *   \param N         NxN neighborhood
 *
 *   \return average pixel Color for the neighborhood
 */
/*======================================================================*/
inline Color averagePixel(CMatrix<Color> const &Img,
                          int x, int y, int N)
{
  Color averagePixel;
  float averagePixelX = 0.0f;
  float averagePixelY = 0.0f;
  float averagePixelZ = 0.0f;

  int lim = N/2;
  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (x+i > 0 && x+i < Img.xSize() &&
          y+j > 0 && y+j < Img.ySize()) {
        averagePixelX += static_cast<float>(Img(x+i, y+j).x);
        averagePixelY += static_cast<float>(Img(x+i, y+j).y);
        averagePixelZ += static_cast<float>(Img(x+i, y+j).z);
      }
    }
  }

  averagePixel.x = averagePixelX/(N*N);
  averagePixel.y = averagePixelY/(N*N);
  averagePixel.z = averagePixelZ/(N*N);
  return averagePixel;
}


/*======================================================================*/
/*! 
 *   Compute difference of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return difference in Colors of a and b
 */
/*======================================================================*/
inline float4 pixelDifference(Color const &a, Color const &b)
{
  float4 pixelDifference;
  pixelDifference.x = static_cast<float>(a.x) - static_cast<float>(b.x);
  pixelDifference.y = static_cast<float>(a.y) - static_cast<float>(b.y);
  pixelDifference.z = static_cast<float>(a.z) - static_cast<float>(b.z);
  return pixelDifference;
}


/*======================================================================*/
/*! 
 *   Compute dot product of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return dot product of a and b
 */
/*======================================================================*/
inline float pixelDotProd(float4 const &a, float4 const &b)
{
  return a.x * b.x +
         a.y * b.y +
         a.z * b.z;
}


/*======================================================================*/
/*! 
 *   Compute Normalized cross-correlation (NCC) 
 *
 *   \param leftImg   Left image
 *   \param rightImg  Right image
 *   \param xl        x position of left image pixel
 *   \param yl        y position of left image pixel
 *   \param xr        x position of right image pixel
 *   \param yr        y position of right image pixel
 *   \param N         NxN neighborhood
 *
 *   \return NCC for pixel neighborhood 
 */
/*======================================================================*/
inline float unaryNCCNeighbor(CMatrix<Color> const &leftImg,
                              CMatrix<Color> const &rightImg,
                              int xl, int yl, int xr, int yr, int N)
{
  float theta = 0.0f;
  int lim = static_cast<int>(N/2);
  float varLeftImg = 0.0f;
  float varRightImg = 0.0f;
  
  Color averagePixelLeftImg;
  Color averagePixelRightImg;
  averagePixelLeftImg = averagePixel(leftImg, xl, yl, N);
  averagePixelRightImg = averagePixel(rightImg, xr, yr, N);

  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (xl+i > 0 && xl+i < leftImg.xSize() &&
          yl+j > 0 && yl+j < leftImg.ySize() &&
          xr+i > 0 && xr+i < rightImg.xSize() &&
          yr+j > 0 && yr+j < rightImg.ySize()) {
        theta += pixelDotProd(
                  pixelDifference(leftImg(xl+i, yl+j), averagePixelLeftImg),            
                  pixelDifference(rightImg(xr+i, yr+j), averagePixelRightImg));
        // Variance of left Image
        varLeftImg += unaryL2Squared(leftImg(xl+i, yl+j),
                                     averagePixelLeftImg);
        // Variance of right Image
        varRightImg += unaryL2Squared(rightImg(xr+i, yr+j),
                                      averagePixelRightImg);
      }
    }
  }

  return theta/std::sqrt(varLeftImg*varRightImg);
}


/*======================================================================*/
/*! 
 *   Potts model for distance of labels a and b. No cost for same label,
 *   constant cost for different labels.
 *
 *   \param a Label of first pixel
 *   \param b Label of second pixel
 *
 *   \return 0 if equal, 1 otherwise
 */
/*======================================================================*/
inline float thetapq(int a, int b)
{
  return (a == b) ? 0.0f : 1.0f;
}


/*======================================================================*/
/*! 
 *   Semi-global Matching between two images
 *
 *   \param result The resulting matrix with disparities
 *   \param leftImg Left Image
 *   \param rightImg Right Image
 *   \param unaryCost Type of unary cost: {pixelWise, L1, L2, NCC}
 *   \param N NxN patch size for L1, L2, NCC unary costs
 *   \param horizontal Horizontal message passing. Set to 1 to activate
 *   \param vertical Vertical message passing. Set to 1 to activate
 *   \param diagonal Diagonal message passing. Set to 1 to activate
 *
 *   \return void, just writes the result in the result matrix
 */
/*======================================================================*/
void sgmCPU(CMatrix<float> &result,
            CMatrix<Color> const &leftImg, CMatrix<Color> const &rightImg,
            int unaryCost, int N,
            bool horizontal, bool vertical, bool diagonal)
{
  /*-----------------------------------------------------------------------
   *  Unary cost computation 
   *-----------------------------------------------------------------------*/
  std::cout << "Precomputing unary costs... \r" << std::flush;
  CTensor<float> unarycosts(
      leftImg.xSize(), leftImg.ySize(), MAX_DISPARITY + 1);
  for (int y = 0; y < leftImg.ySize(); ++y)
  {
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      for (int i = 0; i <= MAX_DISPARITY; ++i)
      {
        if (x - i < 0) unarycosts(x, y, i) = 1.0e9f;
        else {
          switch(unaryCost) {
            case 0:  // Pixel-wise Euclidean distance
              unarycosts(x, y, i) = unaryEuclidean(leftImg(x, y),
                                                   rightImg(x - i, y));
              break;
            case 1:  // NxN L1 distance
              unarycosts(x, y, i) = unaryLxNeighbor(leftImg, rightImg,
                                                     x, y, x - i, y,
                                                     unaryCost, N);
              break;
            case 2:  // NxN L2 distance
              unarycosts(x, y, i) = unaryLxNeighbor(leftImg, rightImg,
                                                     x, y, x - i, y,
                                                     unaryCost, N);
              break;
            case 3:  // NxN NCC
              unarycosts(x, y, i) = -abs(unaryNCCNeighbor(leftImg,
                                                          rightImg,
                                                           x, y, x - i, y,
                                                           N));
              break;
          }
        }
      }
    }
    std::cout << "Precomputing unary costs... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Precomputing unary costs... 100%" << std::endl;

  /*-----------------------------------------------------------------------
   *  Disparity estimation (message passing)
   *-----------------------------------------------------------------------*/

  /* HORIZONTAL (scanline-wise) message Passing */
  std::vector<CMatrix<float> > MpqsHFCube(leftImg.ySize());  // Horizontal Forward 
  std::vector<CMatrix<float> > MpqsHBCube(leftImg.ySize());  // Horizontal Backward
  std::cout << "Computing HORIZONTAL disparities... \r" << std::flush;
  for (int y = 0; y < leftImg.ySize(); ++y)
  {
    /*---------------------------------------------------------------------
     *  Forward pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsHF(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    { 
      MpqsHF(0, j) = 0.0f;
    }
    for (int q = 1; q < leftImg.xSize(); ++q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsHF(q, j) = unarycosts(q - 1, y, 0) + MpqsHF(q - 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q - 1, y, i) + MpqsHF(q - 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsHF(q, j)) {
            MpqsHF(q, j) = cost;
          }
        }
      }
    }
    MpqsHFCube[y] = MpqsHF;

    /*---------------------------------------------------------------------
     *  Backward pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsHB(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsHB(leftImg.xSize() - 1, j) = 0.0f;
    }
    for (int q = leftImg.xSize() - 2; q >= 0; --q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsHB(q, j) = unarycosts(q + 1, y, 0) + MpqsHB(q + 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q + 1, y, i) + MpqsHB(q + 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsHB(q, j)) {
            MpqsHB(q, j) = cost;
          }
        }
      }
    }
    MpqsHBCube[y] = MpqsHB;

    std::cout << "Computing HORIZONTAL disparities... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing HORIZONTAL disparities...100%" << std::endl;
   

  /* VERTICAL (scanline-wise) message Passing */
  std::vector<CMatrix<float> > MpqsVFCube(leftImg.xSize());  // Vertical Forward
  std::vector<CMatrix<float> > MpqsVBCube(leftImg.xSize());  // Vertical Backward
  std::cout << "Computing VERTICAL disparities... \r" << std::flush;
  for (int x = 0; x < leftImg.xSize(); ++x)
  {
    /*---------------------------------------------------------------------
     *  Forward pass (Top to Bottom)
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsVF(leftImg.ySize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsVF(0, j) = 0.0f;
    } 
    for (int q = 1; q < leftImg.ySize(); ++q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsVF(q, j) = unarycosts(x, q - 1, 0) + MpqsVF(q - 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(x, q - 1, i) + MpqsVF(q - 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsVF(q, j)) {
            MpqsVF(q, j) = cost;
          } 
        }
      }
    }
    MpqsVFCube[x] = MpqsVF;

    /*---------------------------------------------------------------------
     *  Backward pass (Bottom to Top)
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsVB(leftImg.ySize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsVB(leftImg.ySize() - 1, j) = 0.0f;
    }
    for (int q = leftImg.ySize() - 2; q >= 0; --q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsVB(q, j) = unarycosts(x, q + 1, 0) + MpqsVB(q + 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(x, q + 1, i) + MpqsVB(q + 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsVB(q, j)) {
            MpqsVB(q, j) = cost;
          } 
        }
      }
    }
    MpqsVBCube[x] = MpqsVB;

    std::cout << "Computing VERTICAL disparities... "
              << static_cast<int>((100.0f * x) / leftImg.xSize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing VERTICAL disparities...100%" << std::endl;


  /* DIAGONAL message Passing */
  std::vector<CMatrix<float> > MpqsDBRCube(leftImg.ySize()); // Diagonal to Bottom Right
  std::vector<CMatrix<float> > MpqsDBLCube(leftImg.ySize()); // Diagonal to Bottom Left
  std::vector<CMatrix<float> > MpqsDTLCube(leftImg.ySize()); // Diagonal to Top Left
  std::vector<CMatrix<float> > MpqsDTRCube(leftImg.ySize()); // Diagonal to Top Right 

  // Initialize top row of disparities matrices
  std::cout << "Computing DIAGONAL disparities... \r" << std::flush;
  CMatrix<float> MpqsDBR(leftImg.xSize(), MAX_DISPARITY + 1);
  CMatrix<float> MpqsDBL(leftImg.xSize(), MAX_DISPARITY + 1);
  CMatrix<float> MpqsDTL(leftImg.xSize(), MAX_DISPARITY + 1);
  CMatrix<float> MpqsDTR(leftImg.xSize(), MAX_DISPARITY + 1);
  for (int x = 0; x < leftImg.xSize(); ++x)
  {
    for(int j = 0; j <= MAX_DISPARITY; ++j) 
    {
      MpqsDBR(0, j) = 0.0f;
      MpqsDBL(0, j) = 0.0f;
      MpqsDTL(0, j) = 0.0f;
      MpqsDTR(0, j) = 0.0f;
    }
  }
  MpqsDBRCube[0] = MpqsDBR;
  MpqsDBLCube[0] = MpqsDBL;
  MpqsDTLCube[leftImg.ySize()-1] = MpqsDTL;
  MpqsDTRCube[leftImg.ySize()-1] = MpqsDTR;

  for (int y = 1; y < leftImg.ySize(); ++y)
  {
    /*---------------------------------------------------------------------
     *  To Bottom Right pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsDBR(leftImg.xSize(), MAX_DISPARITY + 1);
    // Initialize dispaties matrix
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    { 
      MpqsDBR(0, j) = 0.0f;
    }
    for (int q = 1; q < leftImg.xSize(); ++q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsDBR(q, j) = unarycosts(q - 1, y - 1, 0) + MpqsDBRCube[y-1](q - 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q - 1, y - 1, i) + MpqsDBRCube[y-1](q - 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsDBR(q, j)) {
            MpqsDBR(q, j) = cost;
          }
        }
      }
    }
    MpqsDBRCube[y] = MpqsDBR;

    /*---------------------------------------------------------------------
     *  To Bottom Left pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsDBL(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsDBL(leftImg.xSize() - 1, j) = 0.0f;
    }
    for (int q = leftImg.xSize() - 2; q >= 0; --q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsDBL(q, j) = unarycosts(q + 1, y - 1, 0) + MpqsDBLCube[y-1](q + 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q + 1, y - 1, i) + MpqsDBLCube[y-1](q + 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsDBL(q, j)) {
            MpqsDBL(q, j) = cost;
          }
        }
      }
    }
    MpqsDBLCube[y] = MpqsDBL;

    /*---------------------------------------------------------------------
     *  To Top Left pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsDTL(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsDTL(leftImg.xSize() - 1, j) = 0.0f;
    }
    for (int q = leftImg.xSize() - 2; q >= 0; --q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsDTL(q, j) = unarycosts(q + 1, leftImg.ySize() - y, 0)
                        + MpqsDTLCube[leftImg.ySize() - y](q + 1, 0)
                        + LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q + 1, leftImg.ySize(), i) 
                       + MpqsDTLCube[leftImg.ySize() - y](q + 1, i) 
                       + LAMBDA * thetapq(i, j);
          if (cost < MpqsDTL(q, j)) {
            MpqsDTL(q, j) = cost;
          }
        }
      }
    }
    MpqsDTLCube[leftImg.ySize()-y-1] = MpqsDTL;

    /*---------------------------------------------------------------------
     *  To Top Right pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsDTR(leftImg.xSize(), MAX_DISPARITY + 1);
    // Initialize dispaties matrix
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    { 
      MpqsDTR(0, j) = 0.0f;
    }
    for (int q = 1; q < leftImg.xSize(); ++q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsDTR(q, j) = unarycosts(q - 1, leftImg.ySize() - y, 0)
                        + MpqsDTRCube[leftImg.ySize() - y](q - 1, 0)
                        + LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q - 1, leftImg.ySize() - y, i)
                       + MpqsDTRCube[leftImg.ySize() - y](q - 1, i)
                       + LAMBDA * thetapq(i, j);
          if (cost < MpqsDTR(q, j)) {
            MpqsDTR(q, j) = cost;
          }
        }
      }
    }
    MpqsDTRCube[leftImg.ySize()-y-1] = MpqsDTR;

    std::cout << "Computing DIAGONAL disparities... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing DIAGONAL disparities...100%" << std::endl;


  /*---------------------------------------------------------------------
   *  Decision
   *---------------------------------------------------------------------*/
  std::cout << "Computing DECISIONS... \r" << std::flush;
  for (int y = 0; y < leftImg.ySize(); ++y) 
  {
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      int minIndex = 0;
      float minCost = unarycosts(x, y, 0) 
                      + horizontal*(MpqsHFCube[y](x, 0)
                                    + MpqsHBCube[y](x, 0))
                      + vertical*(MpqsVFCube[x](y, 0)
                                  + MpqsVBCube[x](y, 0))
                      + diagonal*(MpqsDBRCube[y](x, 0)
                                 + MpqsDBLCube[y](x, 0)
                                 + MpqsDTLCube[y](x, 0)
                                 + MpqsDTRCube[y](x, 0));
      for (int i = 1; i <= MAX_DISPARITY; ++i)
      {
        float cost = unarycosts(x, y, i) 
                     + horizontal*(MpqsHFCube[y](x, i)
                                  + MpqsHBCube[y](x, i))
                     + vertical*(MpqsVFCube[x](y, i)
                                + MpqsVBCube[x](y, i))
                     + diagonal*(MpqsDBRCube[y](x, i)
                                + MpqsDBLCube[y](x, i)
                                + MpqsDTLCube[y](x, i)
                                + MpqsDTRCube[y](x, i));
        if (cost < minCost)
        {
          minCost = cost;
          minIndex = i;
        }
      }
      result(x, y) = static_cast<float>(minIndex);
    }
    std::cout << "Computing DECISIONS... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing DECISIONS... 100%" << std::endl;  

}


int main(int argc, char** argv)
{
  if (argc < 4)
  {
    std::cerr << "usage: " << argv[0] << " <path to left image> <path to right "
         << "image> <disparity output path>" << std::endl;
    exit(1);
  }

  /*-----------------------------------------------------------------------
   *  Read rectified left and right input image and put them into
   *  Color CMatrices
   *-----------------------------------------------------------------------*/
  CTensor<unsigned char> tmp;
  tmp.readFromPPM(argv[1]);
  CMatrix<Color> leftImg;
  CTensorToColorCMatrix(leftImg, tmp);
  tmp.readFromPPM(argv[2]);
  CMatrix<Color> rightImg;
  CTensorToColorCMatrix(rightImg, tmp);
  
  /*-----------------------------------------------------------------------
   *  Prepare output disparity map
   *-----------------------------------------------------------------------*/
  CMatrix<float> result(leftImg.xSize(), leftImg.ySize());
  
  // Matrix of all options (horizontal, vertical, diagonal)
  int options[4][3][3] = {{{1, 0, 0}, {1, 1, 0}, {1, 1, 1}},
                          {{1, 0, 0}, {1, 1, 0}, {1, 1, 1}},
                          {{1, 0, 0}, {1, 1, 0}, {1, 1, 1}},
                          {{1, 0, 0}, {1, 1, 0}, {1, 1, 1}}
                         };

  int N = 7;  // NxN patches
  timer::start("SGM (CPU)");
  for (int i = 0; i < 4; ++i) {
    for (int j = 0; j < 3; ++j) {
      timer::start("SGM (CPU)");
      int unaryCost = i;
      int horizontal = options[i][j][0];
      int vertical = options[i][j][1];
      int diagonal = options[i][j][2];
      sgmCPU(result, leftImg, rightImg, unaryCost, N,
             horizontal, vertical, diagonal);

      std::string option (NumberToString(i));
      option.append(NumberToString(horizontal));
      option.append(NumberToString(vertical));
      option.append(NumberToString(diagonal));
      std::string resultFile (argv[3]);
      resultFile.insert(0, "-");
      resultFile.insert(0, option); 
      result.writeToFloatFile(resultFile.c_str());

      std::string dispEPE ("../bin/disp-epe ");
      dispEPE.append(resultFile.c_str());
      dispEPE.append(" ");
      dispEPE.append("1-gt.float3");

      std::cout << "Option " << option << std::endl;
      system(dispEPE.c_str());

      std::string floatToPGM ("../bin/float3-to-pgm ");
      floatToPGM.append(resultFile);
      floatToPGM.append(" ");
      floatToPGM.append(resultFile);
      floatToPGM.erase(floatToPGM.length()-6);
      floatToPGM.append("pgm");
      system(floatToPGM.c_str());


      timer::stop("SGM (CPU)");
      timer::printToScreen(std::string(), timer::AUTO_COMMON, timer::ELAPSED_TIME);
    }
  }
  // sgmCPU(result, leftImg, rightImg, 1, 7, 1, 1, 0);
  // Parameters
  // Result Image
  // Left Image
  // Right Image
  // unaryCost {0: PixelWise Euclidean; 1: L1 NxN; 2: L2 NxN; 3: NCC NxN}
  // N NxN patch size
  // horizontal, vertical, diagonal Message Passing

  
  timer::stop("SGM (CPU)");
  timer::printToScreen(std::string(), timer::AUTO_COMMON, timer::ELAPSED_TIME);

  return 0;
}
