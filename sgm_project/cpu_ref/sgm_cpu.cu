#include "hip/hip_runtime.h"
#include "CTensor.h"
#include "timer.h"

#include <cmath>
#include <sstream>
#include <iostream>
#include <iomanip>
#include <map>
#include <string>
#include <stdlib.h>

/************** OPTIONS **************/
// Unary costs and message passing options
static const int unaryCostOption = 3; // (1- Euclidean; 2- L1; 3- L2; 4- NCC)
static const int msgPassingOption = 3; // (1- h; 2- h+v; 3- h+v+d)

// NxN Patch size
static const int N_PATCH = 7;

// Regularization weight
static float const LAMBDA = 100.0f;

// Maximum disparity (number of labels in the message passing algorithm)
static int const MAX_DISPARITY = 50;


/************** CODE **************/
/*-------------------------------------------------------------------------
 *  32Bit RGBA color
 *-------------------------------------------------------------------------*/ 
typedef uchar4 Color;

/*======================================================================*/
/*! 
 *   Convert CTensor to CMatrix of Colors.
 *
 *   \param out The output CMatrix
 *   \param in  The input CTensor
 */
/*======================================================================*/
void CTensorToColorCMatrix(
    CMatrix<Color>& out, const CTensor<unsigned char>& in)
{
  out.setSize(in.xSize(), in.ySize());
  for( int y = 0; y < out.ySize(); ++y )
  for( int x = 0; x < out.xSize(); ++x )
  {
    out(x,y).x = in(x,y,0); // R
    out(x,y).y = in(x,y,1); // G
    out(x,y).z = in(x,y,2); // B
  }
}


/*======================================================================*/
/*! 
 *   Compute squared distance of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L2-distance squared of a and b
 */
/*======================================================================*/
inline float unaryL2Squared(Color const &a, Color const &b)
{
  return (static_cast<float>(a.x) - static_cast<float>(b.x)) *
         (static_cast<float>(a.x) - static_cast<float>(b.x)) +
         (static_cast<float>(a.y) - static_cast<float>(b.y)) *
         (static_cast<float>(a.y) - static_cast<float>(b.y)) +
         (static_cast<float>(a.z) - static_cast<float>(b.z)) *
         (static_cast<float>(a.z) - static_cast<float>(b.z));
}


/*======================================================================*/
/*! 
 *   Compute absolute difference of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L1-distance of a and b
 */
/*======================================================================*/
inline float unaryL1(Color const &a, Color const &b)
{
  return abs(static_cast<float>(a.x) - static_cast<float>(b.x)) +
         abs(static_cast<float>(a.y) - static_cast<float>(b.y)) +
         abs(static_cast<float>(a.z) - static_cast<float>(b.z));
}


/*======================================================================*/
/*! 
 *   Compute euclidean L2 distance of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L2-distance of a and b
 */
/*======================================================================*/
inline float unaryEuclidean(Color const &a, Color const &b)
{
 return std::sqrt(unaryL2Squared(a, b));
}


/*======================================================================*/
/*! 
 *   Compute euclidean Lx-norm for pixel neighborhood 
 *
 *   \param leftImg   Left image
 *   \param rightImg  Right image
 *   \param xl        x position of left image pixel
 *   \param yl        y position of left image pixel
 *   \param xr        x position of right image pixel
 *   \param yr        y position of right image pixel
 *   \param costFunction {1: L1-norm; 2: L2-norm squared} 
 *
 *   \return Lx-norm for pixel neighborhood
 */
/*======================================================================*/
inline float unaryLxNeighbor(CMatrix<Color> const &leftImg,
                             CMatrix<Color> const &rightImg,
                             int xl, int yl, int xr, int yr,
                             int costFunction)
{
  float theta = 0.0f;
  int lim = static_cast<int>(N_PATCH/2);
  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (xl+i >= 0 && xl+i < leftImg.xSize() &&
          yl+j >= 0 && yl+j < leftImg.ySize() &&
          xr+i >= 0 && xr+i < rightImg.xSize() &&
          yr+j >= 0 && yr+j < rightImg.ySize()) {
        switch(costFunction) {
          case 1:
            theta += unaryL1(leftImg(xl+i, yl+j),
                             rightImg(xr+i, yr+j));
            break;
          case 2:
            theta += unaryL2Squared(leftImg(xl+i, yl+j),
                                    rightImg(xr+i, yr+j));
            break;
        }
      }
    }
  }

  return theta;
}


/*======================================================================*/
/*! 
 *   Compute Average pixel of NxN neighborhood 
 *
 *   \param Img       Image
 *   \param x         x position of image pixel
 *   \param y         y position of image pixel
 *
 *   \return average pixel Color for the neighborhood
 */
/*======================================================================*/
inline Color averagePixel(CMatrix<Color> const &Img,
                          int x, int y)
{
  Color averagePixel;
  float averagePixelX = 0.0f;
  float averagePixelY = 0.0f;
  float averagePixelZ = 0.0f;

  int lim = N_PATCH/2;
  int numNeighbors = 0;
  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (x+i >= 0 && x+i < Img.xSize() &&
          y+j >= 0 && y+j < Img.ySize()) {
        averagePixelX += static_cast<float>(Img(x+i, y+j).x);
        averagePixelY += static_cast<float>(Img(x+i, y+j).y);
        averagePixelZ += static_cast<float>(Img(x+i, y+j).z);
        numNeighbors += 1;
      }
    }
  }

  averagePixel.x = averagePixelX/(numNeighbors);
  averagePixel.y = averagePixelY/(numNeighbors);
  averagePixel.z = averagePixelZ/(numNeighbors);
  return averagePixel;
}


/*======================================================================*/
/*! 
 *   Compute difference of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return difference in Colors of a and b
 */
/*======================================================================*/
inline float4 pixelDifference(Color const &a, Color const &b)
{
  float4 pixelDifference;
  pixelDifference.x = static_cast<float>(a.x) - static_cast<float>(b.x);
  pixelDifference.y = static_cast<float>(a.y) - static_cast<float>(b.y);
  pixelDifference.z = static_cast<float>(a.z) - static_cast<float>(b.z);
  return pixelDifference;
}


/*======================================================================*/
/*! 
 *   Compute dot product of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return dot product of a and b
 */
/*======================================================================*/
inline float pixelDotProd(float4 const &a, float4 const &b)
{
  return a.x * b.x +
         a.y * b.y +
         a.z * b.z;
}


/*======================================================================*/
/*! 
 *   Compute Normalized cross-correlation (NCC) 
 *
 *   \param leftImg   Left image
 *   \param rightImg  Right image
 *   \param xl        x position of left image pixel
 *   \param yl        y position of left image pixel
 *   \param xr        x position of right image pixel
 *   \param yr        y position of right image pixel
 *
 *   \return NCC for pixel neighborhood 
 */
/*======================================================================*/
inline float unaryNCCNeighbor(CMatrix<Color> const &leftImg,
                              CMatrix<Color> const &rightImg,
                              int xl, int yl, int xr, int yr)
{
  float theta = 0.0f;
  int lim = static_cast<int>(N_PATCH/2);
  float varLeftImg = 0.0f;
  float varRightImg = 0.0f;
  
  Color averagePixelLeftImg;
  Color averagePixelRightImg;
  averagePixelLeftImg = averagePixel(leftImg, xl, yl);
  averagePixelRightImg = averagePixel(rightImg, xr, yr);

  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      // Check if neighbor is inside the image
      if (xl+i >= 0 && xl+i < leftImg.xSize() && 
          yl+j >= 0 && yl+j < leftImg.ySize() &&
          xr+i >= 0 && xr+i < rightImg.xSize() &&
          yr+j >= 0 && yr+j < rightImg.ySize()) {
        theta += pixelDotProd(
                  pixelDifference(leftImg(xl+i, yl+j), averagePixelLeftImg),            
                  pixelDifference(rightImg(xr+i, yr+j), averagePixelRightImg));
        // Variance of left Image
        varLeftImg += unaryL2Squared(leftImg(xl+i, yl+j),
                                     averagePixelLeftImg);
        // Variance of right Image
        varRightImg += unaryL2Squared(rightImg(xr+i, yr+j),
                                      averagePixelRightImg);
      }
    }
  }

  return theta/std::sqrt(varLeftImg*varRightImg);
}


/*======================================================================*/
/*! 
 *   Potts model for distance of labels a and b. No cost for same label,
 *   constant cost for different labels.
 *
 *   \param a Label of first pixel
 *   \param b Label of second pixel
 *
 *   \return 0 if equal, 1 otherwise
 */
/*======================================================================*/
inline float thetapq(int a, int b)
{
  return (a == b) ? 0.0f : 1.0f;
}


/*======================================================================*/
/*! 
 *   Semi-global Matching between two images
 *
 *   \param result The resulting matrix with disparities
 *   \param leftImg Left Image
 *   \param rightImg Right Image
 *   \param unaryCostOption {pixelWise, L1, L2, NCC}
 *   \param msgPassingOption
 *
 *   \return void, just writes the result in the result matrix
 */
/*======================================================================*/
void sgmCPU(CMatrix<float> &result,
            CMatrix<Color> const &leftImg, CMatrix<Color> const &rightImg,
            int unaryCostOption, int msgPassingOption)
{
  std::cout << std::endl;
  /*-----------------------------------------------------------------------
   *  Unary cost computation 
   *-----------------------------------------------------------------------*/
  std::cout << "Precomputing unary costs... \r" << std::flush;
  CTensor<float> unarycosts(
      leftImg.xSize(), leftImg.ySize(), MAX_DISPARITY + 1);
  for (int y = 0; y < leftImg.ySize(); ++y)
  {
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      for (int i = 0; i <= MAX_DISPARITY; ++i)
      {
        if (x - i < 0) unarycosts(x, y, i) = 1.0e9f;
        else {
          switch(unaryCostOption) {
            case 1:  // Pixel-wise Euclidean distance
              unarycosts(x, y, i) = unaryEuclidean(leftImg(x, y),
                                                   rightImg(x - i, y));
              break;
            case 2:  // NxN L1 distance
              unarycosts(x, y, i) = unaryLxNeighbor(leftImg, rightImg,
                                                     x, y, x - i, y, 1);
              break;
            case 3:  // NxN L2 distance
              unarycosts(x, y, i) = unaryLxNeighbor(leftImg, rightImg,
                                                     x, y, x - i, y, 2);
              break;
            case 4:  // NxN NCC
              unarycosts(x, y, i) = -abs(unaryNCCNeighbor(leftImg, rightImg,
                                                          x, y, x - i, y));
              break;
          }
        }
      }
    }
    std::cout << "Precomputing unary costs... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Precomputing unary costs... 100%" << std::endl;

  /*-----------------------------------------------------------------------
   *  Disparity estimation (message passing)
   *-----------------------------------------------------------------------*/

  /* HORIZONTAL (scanline-wise) message Passing */
  std::vector<CMatrix<float> > MpqsHFCube(leftImg.ySize());  // Horizontal Forward 
  std::vector<CMatrix<float> > MpqsHBCube(leftImg.ySize());  // Horizontal Backward
  if (msgPassingOption == 1 || msgPassingOption == 2 || msgPassingOption == 3)
  {
        std::cout << "Computing HORIZONTAL disparities... \r" << std::flush;
    for (int y = 0; y < leftImg.ySize(); ++y)
    {
      /*---------------------------------------------------------------------
       *  Forward pass
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsHF(leftImg.xSize(), MAX_DISPARITY + 1);
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      { 
        MpqsHF(0, j) = 0.0f;
      }
      for (int q = 1; q < leftImg.xSize(); ++q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsHF(q, j) = unarycosts(q - 1, y, 0) + MpqsHF(q - 1, 0) +
                LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(q - 1, y, i) + MpqsHF(q - 1, i) +
                LAMBDA * thetapq(i, j);
            if (cost < MpqsHF(q, j)) {
              MpqsHF(q, j) = cost;
            }
          }
        }
      }
      MpqsHFCube[y] = MpqsHF;

      /*---------------------------------------------------------------------
       *  Backward pass
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsHB(leftImg.xSize(), MAX_DISPARITY + 1);
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsHB(leftImg.xSize() - 1, j) = 0.0f;
      }
      for (int q = leftImg.xSize() - 2; q >= 0; --q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsHB(q, j) = unarycosts(q + 1, y, 0) + MpqsHB(q + 1, 0) +
                LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(q + 1, y, i) + MpqsHB(q + 1, i) +
                LAMBDA * thetapq(i, j);
            if (cost < MpqsHB(q, j)) {
              MpqsHB(q, j) = cost;
            }
          }
        }
      }
      MpqsHBCube[y] = MpqsHB;

      std::cout << "Computing HORIZONTAL disparities... "
                << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
                << std::flush;
    }
  }
    std::cout << "Computing HORIZONTAL disparities...100%" << std::endl;
   

  /* VERTICAL (scanline-wise) message Passing */
  std::vector<CMatrix<float> > MpqsVFCube(leftImg.xSize());  // Vertical Forward
  std::vector<CMatrix<float> > MpqsVBCube(leftImg.xSize());  // Vertical Backward
  if (msgPassingOption == 2 || msgPassingOption == 3)
  {
    std::cout << "Computing VERTICAL disparities... \r" << std::flush;
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      /*---------------------------------------------------------------------
       *  Forward pass (Top to Bottom)
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsVF(leftImg.ySize(), MAX_DISPARITY + 1);
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsVF(0, j) = 0.0f;
      } 
      for (int q = 1; q < leftImg.ySize(); ++q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsVF(q, j) = unarycosts(x, q - 1, 0) + MpqsVF(q - 1, 0) +
                LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(x, q - 1, i) + MpqsVF(q - 1, i) +
                LAMBDA * thetapq(i, j);
            if (cost < MpqsVF(q, j)) {
              MpqsVF(q, j) = cost;
            } 
          }
        }
      }
      MpqsVFCube[x] = MpqsVF;

      /*---------------------------------------------------------------------
       *  Backward pass (Bottom to Top)
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsVB(leftImg.ySize(), MAX_DISPARITY + 1);
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsVB(leftImg.ySize() - 1, j) = 0.0f;
      }
      for (int q = leftImg.ySize() - 2; q >= 0; --q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsVB(q, j) = unarycosts(x, q + 1, 0) + MpqsVB(q + 1, 0) +
                LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(x, q + 1, i) + MpqsVB(q + 1, i) +
                LAMBDA * thetapq(i, j);
            if (cost < MpqsVB(q, j)) {
              MpqsVB(q, j) = cost;
            } 
          }
        }
      }
      MpqsVBCube[x] = MpqsVB;

      std::cout << "Computing VERTICAL disparities... "
                << static_cast<int>((100.0f * x) / leftImg.xSize()) << "% \r"
                << std::flush;
    }
    std::cout << "Computing VERTICAL disparities...100%" << std::endl;
  }

  /* DIAGONAL message Passing */
  std::vector<CMatrix<float> > MpqsDBRCube(leftImg.ySize()); // Diagonal to Bottom Right
  std::vector<CMatrix<float> > MpqsDBLCube(leftImg.ySize()); // Diagonal to Bottom Left
  std::vector<CMatrix<float> > MpqsDTLCube(leftImg.ySize()); // Diagonal to Top Left
  std::vector<CMatrix<float> > MpqsDTRCube(leftImg.ySize()); // Diagonal to Top Right 
  if (msgPassingOption == 3)
  {
    // Initialize top row of disparities matrices
    std::cout << "Computing DIAGONAL disparities... \r" << std::flush;
    CMatrix<float> MpqsDBR(leftImg.xSize(), MAX_DISPARITY + 1);
    CMatrix<float> MpqsDBL(leftImg.xSize(), MAX_DISPARITY + 1);
    CMatrix<float> MpqsDTL(leftImg.xSize(), MAX_DISPARITY + 1);
    CMatrix<float> MpqsDTR(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      for(int j = 0; j <= MAX_DISPARITY; ++j) 
      {
        MpqsDBR(0, j) = 0.0f;
        MpqsDBL(0, j) = 0.0f;
        MpqsDTL(0, j) = 0.0f;
        MpqsDTR(0, j) = 0.0f;
      }
    }
    MpqsDBRCube[0] = MpqsDBR;
    MpqsDBLCube[0] = MpqsDBL;
    MpqsDTLCube[leftImg.ySize()-1] = MpqsDTL;
    MpqsDTRCube[leftImg.ySize()-1] = MpqsDTR;

    for (int y = 1; y < leftImg.ySize(); ++y)
    {
      /*---------------------------------------------------------------------
       *  To Bottom Right pass
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsDBR(leftImg.xSize(), MAX_DISPARITY + 1);
      // Initialize dispaties matrix
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      { 
        MpqsDBR(0, j) = 0.0f;
      }
      for (int q = 1; q < leftImg.xSize(); ++q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsDBR(q, j) = unarycosts(q - 1, y - 1, 0) + MpqsDBRCube[y-1](q - 1, 0) +
                LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(q - 1, y - 1, i) + MpqsDBRCube[y-1](q - 1, i) +
                LAMBDA * thetapq(i, j);
            if (cost < MpqsDBR(q, j)) {
              MpqsDBR(q, j) = cost;
            }
          }
        }
      }
      MpqsDBRCube[y] = MpqsDBR;

      /*---------------------------------------------------------------------
       *  To Bottom Left pass
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsDBL(leftImg.xSize(), MAX_DISPARITY + 1);
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsDBL(leftImg.xSize() - 1, j) = 0.0f;
      }
      for (int q = leftImg.xSize() - 2; q >= 0; --q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsDBL(q, j) = unarycosts(q + 1, y - 1, 0) + MpqsDBLCube[y-1](q + 1, 0) +
                LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(q + 1, y - 1, i) + MpqsDBLCube[y-1](q + 1, i) +
                LAMBDA * thetapq(i, j);
            if (cost < MpqsDBL(q, j)) {
              MpqsDBL(q, j) = cost;
            }
          }
        }
      }
      MpqsDBLCube[y] = MpqsDBL;

      /*---------------------------------------------------------------------
       *  To Top Left pass
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsDTL(leftImg.xSize(), MAX_DISPARITY + 1);
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsDTL(leftImg.xSize() - 1, j) = 0.0f;
      }
      for (int q = leftImg.xSize() - 2; q >= 0; --q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsDTL(q, j) = unarycosts(q + 1, leftImg.ySize() - y, 0)
                          + MpqsDTLCube[leftImg.ySize() - y](q + 1, 0)
                          + LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(q + 1, leftImg.ySize(), i) 
                         + MpqsDTLCube[leftImg.ySize() - y](q + 1, i) 
                         + LAMBDA * thetapq(i, j);
            if (cost < MpqsDTL(q, j)) {
              MpqsDTL(q, j) = cost;
            }
          }
        }
      }
      MpqsDTLCube[leftImg.ySize()-y-1] = MpqsDTL;

      /*---------------------------------------------------------------------
       *  To Top Right pass
       *---------------------------------------------------------------------*/
      CMatrix<float> MpqsDTR(leftImg.xSize(), MAX_DISPARITY + 1);
      // Initialize dispaties matrix
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      { 
        MpqsDTR(0, j) = 0.0f;
      }
      for (int q = 1; q < leftImg.xSize(); ++q)
      {
        for (int j = 0; j <= MAX_DISPARITY; ++j)
        {
          MpqsDTR(q, j) = unarycosts(q - 1, leftImg.ySize() - y, 0)
                          + MpqsDTRCube[leftImg.ySize() - y](q - 1, 0)
                          + LAMBDA * thetapq(0, j);
          for (int i = 1; i <= MAX_DISPARITY; ++i)
          {
            float cost = unarycosts(q - 1, leftImg.ySize() - y, i)
                         + MpqsDTRCube[leftImg.ySize() - y](q - 1, i)
                         + LAMBDA * thetapq(i, j);
            if (cost < MpqsDTR(q, j)) {
              MpqsDTR(q, j) = cost;
            }
          }
        }
      }
      MpqsDTRCube[leftImg.ySize()-y-1] = MpqsDTR;

      std::cout << "Computing DIAGONAL disparities... "
                << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
                << std::flush;
    }
    std::cout << "Computing DIAGONAL disparities...100%" << std::endl;
  }

  /*---------------------------------------------------------------------
   *  Decision
   *---------------------------------------------------------------------*/
  std::cout << "Computing DECISIONS... \r" << std::flush;
  for (int y = 0; y < leftImg.ySize(); ++y) 
  {
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      int minIndex = 0;
      float minCost = unarycosts(x, y, 0);
      if (msgPassingOption == 1 || msgPassingOption == 2 || msgPassingOption == 3) {
        minCost += MpqsHFCube[y](x, 0) + MpqsHBCube[y](x, 0);
      }
      if (msgPassingOption == 2 || msgPassingOption == 3) {
        minCost += MpqsVFCube[x](y, 0) + MpqsVBCube[x](y, 0);
      }
      if (msgPassingOption == 3) {
        minCost += MpqsDBRCube[y](x, 0) + MpqsDBLCube[y](x, 0)
                   + MpqsDTLCube[y](x, 0) + MpqsDTRCube[y](x, 0);
      }

      for (int i = 1; i <= MAX_DISPARITY; ++i)
      {
        float cost = unarycosts(x, y, i);
        if (msgPassingOption == 1 || msgPassingOption == 2 || msgPassingOption == 3) {
          cost += MpqsHFCube[y](x, i) + MpqsHBCube[y](x, i);
        }
        if (msgPassingOption == 2 || msgPassingOption == 3) {
          cost += MpqsVFCube[x](y, i) + MpqsVBCube[x](y, i);
        }
        if (msgPassingOption == 3) {
          cost += MpqsDBRCube[y](x, i) + MpqsDBLCube[y](x, i)
                  + MpqsDTLCube[y](x, i) + MpqsDTRCube[y](x, i);
        }

        if (cost < minCost)
        {
          minCost = cost;
          minIndex = i;
        }
      }
      result(x, y) = static_cast<float>(minIndex);
    }
    std::cout << "Computing DECISIONS... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing DECISIONS... 100%" << std::endl;   
}


int main(int argc, char** argv)
{
  if (argc < 4)
  {
    std::cerr << "usage: " << argv[0] << " <path to left image> <path to right "
         << "image> <disparity output path>" << std::endl;
    exit(1);
  }
  std::string outputFile(argv[3]);
  // unaryCosts map
  std::map<int, std::string> unaryCostsMap;
  unaryCostsMap.insert(std::make_pair(1, "PixelWise Euclidean"));
  unaryCostsMap.insert(std::make_pair(2, "L1"));
  unaryCostsMap.insert(std::make_pair(3, "L2"));
  unaryCostsMap.insert(std::make_pair(4, "NCC"));
  // msgPassOption map (horizontal, vertical, diagonal)
  std::map<int, std::string> msgPassOptionMap;
  msgPassOptionMap.insert(std::make_pair(1, "horizontal"));
  msgPassOptionMap.insert(std::make_pair(2, "horizontal + vertical"));
  msgPassOptionMap.insert(std::make_pair(3, "horizontal + vertical + diagonal"));

  // Output unary costs and message passing option
  std::cout << std::endl;
  if (unaryCostOption == 1) {
    std::cout << "Unary cost : " << unaryCostsMap[unaryCostOption] << std::endl;
  } else {
    std::cout << "Unary cost: " << unaryCostsMap[unaryCostOption];
    std::cout << " " << N_PATCH << "x" << N_PATCH << " patches" << std::endl;
  }
  std::cout << "Msg Passing: " << msgPassOptionMap[msgPassingOption] << std::endl;

  /*-----------------------------------------------------------------------
   *  Read rectified left and right input image and put them into
   *  Color CMatrices
   *-----------------------------------------------------------------------*/
  CTensor<unsigned char> tmp;
  tmp.readFromPPM(argv[1]);
  CMatrix<Color> leftImg;
  CTensorToColorCMatrix(leftImg, tmp);
  tmp.readFromPPM(argv[2]);
  CMatrix<Color> rightImg;
  CTensorToColorCMatrix(rightImg, tmp);
  
  /*-----------------------------------------------------------------------
   *  Prepare output disparity map
   *-----------------------------------------------------------------------*/
  CMatrix<float> result(leftImg.xSize(), leftImg.ySize());
  
  // Compute Semi-global matching
  timer::start("SGM (CPU)");
  sgmCPU(result, leftImg, rightImg, unaryCostOption, msgPassingOption);
  timer::stop("SGM (CPU)");
  timer::printToScreen(std::string(), timer::AUTO_COMMON, timer::ELAPSED_TIME);

  /*---------------------------------------------------------------------
   *  Write results to output file 
   *---------------------------------------------------------------------*/
  result.writeToFloatFile(outputFile.c_str());

  return 0;
}
