#include "hip/hip_runtime.h"
#include "CTensor.h"
#include "timer.h"

#include <cmath>

/*-------------------------------------------------------------------------
 *  32Bit RGBA color
 *-------------------------------------------------------------------------*/ 
typedef uchar4 Color;

/*-------------------------------------------------------------------------
 *  Regularization weight
 *-------------------------------------------------------------------------*/
static float const LAMBDA = 100.0f;

/*-------------------------------------------------------------------------
 *  Maximum disparity (number of labels in the message passing algorithm)
 *-------------------------------------------------------------------------*/ 
static int const MAX_DISPARITY = 50;

/*======================================================================*/
/*! 
 *   Convert CTensor to CMatrix of Colors.
 *
 *   \param out The output CMatrix
 *   \param in  The input CTensor
 */
/*======================================================================*/
void CTensorToColorCMatrix(
    CMatrix<Color>& out, const CTensor<unsigned char>& in)
{
  out.setSize(in.xSize(), in.ySize());
  for( int y = 0; y < out.ySize(); ++y )
  for( int x = 0; x < out.xSize(); ++x )
  {
    out(x,y).x = in(x,y,0); // R
    out(x,y).y = in(x,y,1); // G
    out(x,y).z = in(x,y,2); // B
  }
}


/*======================================================================*/
/*! 
 *   Compute squared distance of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L2-distance squared of a and b
 */
/*======================================================================*/
inline float unaryL2Squared(Color const &a, Color const &b)
{
  return (static_cast<float>(a.x) - static_cast<float>(b.x)) *
         (static_cast<float>(a.x) - static_cast<float>(b.x)) +
         (static_cast<float>(a.y) - static_cast<float>(b.y)) *
         (static_cast<float>(a.y) - static_cast<float>(b.y)) +
         (static_cast<float>(a.z) - static_cast<float>(b.z)) *
         (static_cast<float>(a.z) - static_cast<float>(b.z));
}


/*======================================================================*/
/*! 
 *   Compute absolute difference of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L1-distance of a and b
 */
/*======================================================================*/
inline float unaryL1(Color const &a, Color const &b)
{
  return abs(static_cast<float>(a.x) - static_cast<float>(b.x)) +
         abs(static_cast<float>(a.y) - static_cast<float>(b.y)) +
         abs(static_cast<float>(a.z) - static_cast<float>(b.z));
}


/*======================================================================*/
/*! 
 *   Compute euclidean L2 distance of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return L2-distance of a and b
 */
/*======================================================================*/
inline float unaryEuclidean(Color const &a, Color const &b)
{
 return std::sqrt(unaryL2Squared(a, b));
}


/*======================================================================*/
/*! 
 *   Compute euclidean Lx-norm for pixel neighborhood 
 *
 *   \param leftImg   Left image
 *   \param rightImg  Right image
 *   \param xl        x position of left image pixel
 *   \param yl        y position of left image pixel
 *   \param xr        x position of right image pixel
 *   \param yr        y position of right image pixel
 *   \param N         NxN neighborhood
 *   \param option    1: L1-norm; 2: L2-norm squared
 *
 *   \return Lx-norm for pixel neighborhood
 */
/*======================================================================*/
inline float unaryLxNeighbor(CMatrix<Color> const &leftImg,
                             CMatrix<Color> const &rightImg,
                             int xl, int yl, int xr, int yr, int N,
                             int option)
{
  float theta = 0.0f;
  int lim = static_cast<int>(N/2);
  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
        if (xl+i > 0 && xl+i < leftImg.xSize() &&
            yl+j > 0 && yl+j < leftImg.ySize() &&
            xr+i > 0 && xr+i < rightImg.xSize() &&
            yr+j > 0 && yr+j < rightImg.ySize()) {
          if (option == 1)
            theta += unaryL1(leftImg(xl+i, yl+j), rightImg(xr+i, yr+j));
          if (option == 2)
            theta += unaryL2Squared(leftImg(xl+i, yl+j), rightImg(xr+i, yr+j));
        }
    }
  }

  return theta;
}


/*======================================================================*/
/*! 
 *   Compute Average pixel of NxN neighborhood 
 *
 *   \param Img       Image
 *   \param x         x position of image pixel
 *   \param y         y position of image pixel
 *   \param N         NxN neighborhood
 *
 *   \return average pixel Color for the neighborhood
 */
/*======================================================================*/
inline Color averagePixel(CMatrix<Color> const &Img,
                          int x, int y, int N)
{
  Color averagePixel;
  float averagePixelX = 0.0f;
  float averagePixelY = 0.0f;
  float averagePixelZ = 0.0f;

  int lim = N/2;
  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (x+i > 0 && x+i < Img.xSize() &&
          y+j > 0 && y+j < Img.ySize()) {
        averagePixelX += static_cast<float>(Img(x+i, y+j).x);
        averagePixelY += static_cast<float>(Img(x+i, y+j).y);
        averagePixelZ += static_cast<float>(Img(x+i, y+j).z);
      }
    }
  }

  averagePixel.x = averagePixelX/(N*N);
  averagePixel.y = averagePixelY/(N*N);
  averagePixel.z = averagePixelZ/(N*N);
  return averagePixel;
}


/*======================================================================*/
/*! 
 *   Compute difference of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return difference in Colors of a and b
 */
/*======================================================================*/
inline float4 pixelDifference(Color const &a, Color const &b)
{
  float4 pixelDifference;
  pixelDifference.x = static_cast<float>(a.x) - static_cast<float>(b.x);
  pixelDifference.y = static_cast<float>(a.y) - static_cast<float>(b.y);
  pixelDifference.z = static_cast<float>(a.z) - static_cast<float>(b.z);
  return pixelDifference;
}


/*======================================================================*/
/*! 
 *   Compute dot product of given pixels.
 *
 *   \param a The first pixel
 *   \param b The second pixel
 *
 *   \return dot product of a and b
 */
/*======================================================================*/
inline float pixelDotProd(float4 const &a, float4 const &b)
{
  return a.x * b.x +
         a.y * b.y +
         a.z * b.z;
}


/*======================================================================*/
/*! 
 *   Compute Normalized cross-correlation (NCC) 
 *
 *   \param leftImg   Left image
 *   \param rightImg  Right image
 *   \param xl        x position of left image pixel
 *   \param yl        y position of left image pixel
 *   \param xr        x position of right image pixel
 *   \param yr        y position of right image pixel
 *   \param N         NxN neighborhood
 *
 *   \return NCC for pixel neighborhood 
 */
/*======================================================================*/
inline float unaryNCCNeighbor(CMatrix<Color> const &leftImg,
                              CMatrix<Color> const &rightImg,
                              int xl, int yl, int xr, int yr, int N)
{
  float theta = 0.0f;
  int lim = static_cast<int>(N/2);
  float varLeftImg = 0.0f;
  float varRightImg = 0.0f;
  
  Color averagePixelLeftImg;
  Color averagePixelRightImg;
  averagePixelLeftImg = averagePixel(leftImg, xl, yl, N);
  averagePixelRightImg = averagePixel(rightImg, xr, yr, N);

  for (int j = -lim; j < lim; ++j) {
    for (int i = -lim; i < lim; ++i) {
      if (xl+i > 0 && xl+i < leftImg.xSize() &&
          yl+j > 0 && yl+j < leftImg.ySize() &&
          xr+i > 0 && xr+i < rightImg.xSize() &&
          yr+j > 0 && yr+j < rightImg.ySize()) {
        theta += pixelDotProd(
                  pixelDifference(leftImg(xl+i, yl+j), averagePixelLeftImg),            
                  pixelDifference(rightImg(xr+i, yr+j), averagePixelRightImg));
        // Variance of left Image
        varLeftImg += unaryL2Squared(leftImg(xl+i, yl+j),
                                     averagePixelLeftImg);
        // Variance of right Image
        varRightImg += unaryL2Squared(rightImg(xr+i, yr+j),
                                      averagePixelRightImg);
      }
    }
  }

  return theta/std::sqrt(varLeftImg*varRightImg);
}


/*======================================================================*/
/*! 
 *   Potts model for distance of labels a and b. No cost for same label,
 *   constant cost for different labels.
 *
 *   \param a Label of first pixel
 *   \param b Label of second pixel
 *
 *   \return 0 if equal, 1 otherwise
 */
/*======================================================================*/
inline float thetapq(int a, int b)
{
  return (a == b) ? 0.0f : 1.0f;
}

void sgmCPU(CMatrix<float> &result,
            CMatrix<Color> const &leftImg, CMatrix<Color> const &rightImg)
{
  /*-----------------------------------------------------------------------
   *  Unary cost computation (Currently simple L2 color distance)
   *
   *  ** ToDo: Extend this to block-matching using L2, L1 and NCC metrics
   *  **       optional: try different block shapes (and shape combinations)
   *-----------------------------------------------------------------------*/
  std::cout << "Precomputing unary costs... \r" << std::flush;
  CTensor<float> unarycosts(
      leftImg.xSize(), leftImg.ySize(), MAX_DISPARITY + 1);
  for (int y = 0; y < leftImg.ySize(); ++y)
  {
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      for (int i = 0; i <= MAX_DISPARITY; ++i)
      {
        if (x - i < 0) unarycosts(x, y, i) = 1.0e9f;
        // else unarycosts(x, y, i) = unaryEuclidean(leftImg(x, y), rightImg(x - i, y));
        else unarycosts(x, y, i) = unaryLxNeighbor(leftImg, rightImg, x, y, x - i, y, 7, 2);
        // else unarycosts(x, y, i) = -abs(unaryNCCNeighbor(leftImg, rightImg, x, y, x - i, y, 7));
      }
    }
    std::cout << "Precomputing unary costs... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Precomputing unary costs... 100%" << std::endl;

  /*-----------------------------------------------------------------------
   *  Disparity estimation (Scanline-wise message passing)
   *  ** ToDo: for Bachelors: Extend this to horizontal and vertical message
   *  **                      passing
   *  **       for Masters: Extend this to 8 directions
   *  **                    (i.e. 0,22.5,45,67.5,90,112.5,135,157.5 degrees)
   *-----------------------------------------------------------------------*/

  /* HORIZONTAL (scanline-wise) message Passing */
  std::vector<CMatrix<float> > MpqsF_(leftImg.ySize());
  std::vector<CMatrix<float> > MpqsB_(leftImg.ySize());

  std::cout << "Computing HORIZONTAL disparities... \r" << std::flush;
  for (int y = 0; y < leftImg.ySize(); ++y)
  {
    /*---------------------------------------------------------------------
     *  Forward pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsF(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    { 
      MpqsF(0, j) = 0.0f;
    }
    for (int q = 1; q < leftImg.xSize(); ++q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsF(q, j) = unarycosts(q - 1, y, 0) + MpqsF(q - 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q - 1, y, i) + MpqsF(q - 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsF(q, j)) {
            MpqsF(q, j) = cost;
          }
        }
      }
    }
    MpqsF_[y] = MpqsF;

    /*---------------------------------------------------------------------
     *  Backward pass
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsB(leftImg.xSize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsB(leftImg.xSize() - 1, j) = 0.0f;
    }
    for (int q = leftImg.xSize() - 2; q >= 0; --q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsB(q, j) = unarycosts(q + 1, y, 0) + MpqsB(q + 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(q + 1, y, i) + MpqsB(q + 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsB(q, j)) {
            MpqsB(q, j) = cost;
          }
        }
      }
    }
    MpqsB_[y] = MpqsB;
    std::cout << "Computing HORIZONTAL disparities... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing HORIZONTAL disparities...100%" << std::endl;

    
  /* VERTICAL (scanline-wise) message Passing */
  std::vector<CMatrix<float> > MpqsVF_(leftImg.xSize());
  std::vector<CMatrix<float> > MpqsVB_(leftImg.xSize());

  std::cout << "Computing VERTICAL disparities... \r" << std::flush;
  for (int x = 0; x < leftImg.xSize(); ++x)
  {
    /*---------------------------------------------------------------------
     *  Forward pass (Top to Bottom)
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsVF(leftImg.ySize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsVF(0, j) = 0.0f;
    } 
    for (int q = 1; q < leftImg.ySize(); ++q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsVF(q, j) = unarycosts(x, q - 1, 0) + MpqsVF(q - 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(x, q - 1, i) + MpqsVF(q - 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsVF(q, j)) {
            MpqsVF(q, j) = cost;
          } 
        }
      }
    }
    MpqsVF_[x] = MpqsVF;

    /*---------------------------------------------------------------------
     *  Backward pass (Bottom to Top)
     *---------------------------------------------------------------------*/
    CMatrix<float> MpqsVB(leftImg.ySize(), MAX_DISPARITY + 1);
    for (int j = 0; j <= MAX_DISPARITY; ++j)
    {
      MpqsVB(leftImg.ySize() - 1, j) = 0.0f;
    }
    for (int q = leftImg.ySize() - 2; q >= 0; --q)
    {
      for (int j = 0; j <= MAX_DISPARITY; ++j)
      {
        MpqsVB(q, j) = unarycosts(x, q + 1, 0) + MpqsVB(q + 1, 0) +
              LAMBDA * thetapq(0, j);
        for (int i = 1; i <= MAX_DISPARITY; ++i)
        {
          float cost = unarycosts(x, q + 1, i) + MpqsVB(q + 1, i) +
              LAMBDA * thetapq(i, j);
          if (cost < MpqsVB(q, j)) {
            MpqsVB(q, j) = cost;
          } 
        }
      }
    }
    MpqsVB_[x] = MpqsVB;
    std::cout << "Computing VERTICAL disparities... "
              << static_cast<int>((100.0f * x) / leftImg.xSize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing VERTICAL disparities...100%" << std::endl;


  /*---------------------------------------------------------------------
   *  Decision
   *---------------------------------------------------------------------*/
  std::cout << "Computing DECISIONS... \r" << std::flush;
  for (int y = 0; y < leftImg.ySize(); ++y) 
  {
    for (int x = 0; x < leftImg.xSize(); ++x)
    {
      int minIndex = 0;
      float minCost = unarycosts(x, y, 0) 
                      + MpqsF_[y](x, 0)
                      + MpqsB_[y](x, 0)
                      + MpqsVF_[x](y, 0)
                      + MpqsVB_[x](y, 0);
      for (int i = 1; i <= MAX_DISPARITY; ++i)
      {
        float cost = unarycosts(x, y, i) 
                     + MpqsF_[y](x, i)
                     + MpqsB_[y](x, i)
                     + MpqsVF_[x](y, i)
                     + MpqsVB_[x](y, i);
        if (cost < minCost)
        {
          minCost = cost;
          minIndex = i;
        }
      }
      result(x, y) = static_cast<float>(minIndex);
    }
    std::cout << "Computing DECISIONS... "
              << static_cast<int>((100.0f * y) / leftImg.ySize()) << "% \r"
              << std::flush;
  }
  std::cout << "Computing DECISIONS... 100%" << std::endl;  

}


int main(int argc, char** argv)
{
  if (argc < 4)
  {
    std::cerr << "usage: " << argv[0] << " <path to left image> <path to right "
         << "image> <disparity output path>" << std::endl;
    exit(1);
  }

  /*-----------------------------------------------------------------------
   *  Read rectified left and right input image and put them into
   *  Color CMatrices
   *-----------------------------------------------------------------------*/
  CTensor<unsigned char> tmp;
  tmp.readFromPPM(argv[1]);
  CMatrix<Color> leftImg;
  CTensorToColorCMatrix(leftImg, tmp);
  tmp.readFromPPM(argv[2]);
  CMatrix<Color> rightImg;
  CTensorToColorCMatrix(rightImg, tmp);
  
  /*-----------------------------------------------------------------------
   *  Prepare output disparity map
   *-----------------------------------------------------------------------*/
  CMatrix<float> result(leftImg.xSize(), leftImg.ySize());
  
  timer::start("SGM (CPU)");
  sgmCPU(result, leftImg, rightImg);
  timer::stop("SGM (CPU)");

  result.writeToFloatFile(argv[3]);

  timer::printToScreen(
      std::string(), timer::AUTO_COMMON, timer::ELAPSED_TIME);

  return 0;
}
